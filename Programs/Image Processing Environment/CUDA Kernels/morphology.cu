#include "hip/hip_runtime.h"
#include <stdio.h>


#define Erode 0
#define Dilate 1

__global__ void morphology(int* image, const int Width, const int Height, const int Operation, const int Radius)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if (i >= Width || j >= Height) return;
	int index = j * Width + i;
	int value = (Operation == Erode) ? 0 : 255;
	for (int h = (i - Radius); h < (i + Radius); h++)
	{
		for (int w = (j - Radius); w < (j + Radius); w++)
		{
			if (w >= 0 && w < Width && h >= 0 && h < Height)
			{
				int maskIndex = h * Width + w;
				if (image[maskIndex] == value)
				{
					image[index] = value;
					w = j + Radius;
					h = i + Radius;
				}
			}			
		}
	}
}
 
int main()
{
    return 0;
}