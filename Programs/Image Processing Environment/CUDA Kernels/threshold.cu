#include "hip/hip_runtime.h"
#include <stdio.h>


__global__ void threshold(int* image, const int Width, const int Height, const int Threshold)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if (i >= Width || j >= Height) return;
	int index = j * Width + i;
	if (image[index] >= Threshold)
	{
		image[index] = 255;
	}
	else
	{
		image[index] = 0;
	}
}